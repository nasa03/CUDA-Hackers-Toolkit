#include "hip/hip_runtime.h"
#include <iostream>

// Define hello world kernel
__global__ void HelloWorldKernel() {
    printf("Hello World № %d!\n", threadIdx.x * gridDim.x);
}

int main() {
    // Define execution configuration variables
    int numBlocksPerGrid = 1, numThreadsPerBlock = 32;

    // Launch hello world kernel on device
    HelloWorldKernel << <numBlocksPerGrid, numThreadsPerBlock >> > ();

    // Wait for the device to finish
    hipDeviceSynchronize();

    // Check for errors
    int exitStatus = EXIT_SUCCESS;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << '\n';
        exitStatus = EXIT_FAILURE;
    }

    return exitStatus;
}