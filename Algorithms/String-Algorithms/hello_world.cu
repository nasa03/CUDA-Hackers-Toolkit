#include "hip/hip_runtime.h"
#include <iostream>

// Define execution configuration variables
constexpr int numBlocksPerGrid = 1;
constexpr int numThreadsPerBlock = 64;

// Define hello world kernel
__global__ void HelloWorldKernel() {
    printf("Hello World № %d!\n", threadIdx.x * gridDim.x);
}

int main() {
    // HOST EXECUTION

    // Declare host clock variables
	float elapsedTimeHost;
    clock_t startTimeHost, stopTimeHost;

    // Start host clock
    startTimeHost = clock();

    // Launch execution on host
    for (int i = 0; i < numThreadsPerBlock; ++i) {
        std::cout << "Hello World №" << i << "!\n";
    }

    // Stop host clock
    stopTimeHost = clock();
    elapsedTimeHost = (float) ((stopTimeHost) - (startTimeHost));
	printf("Host Elapsed Time: %f ms\n", elapsedTimeHost);
    
    // DEVICE EXECUTION

    // Declare device clock variables
	float elapsedTimeDevice;
    hipEvent_t startTimeDevice, stopTimeDevice;

    // Start device clock
    hipEventCreate(&startTimeDevice);
	hipEventRecord(startTimeDevice, 0);

    // Launch hello world kernel on device
    HelloWorldNonSharedMemoryKernel <<<numBlocksPerGrid, numThreadsPerBlock>>> ();

    // Wait for the device to finish computing
    hipDeviceSynchronize();

    // Stop device clock
    hipEventCreate(&stopTimeDevice);
	hipEventRecord(stopTimeDevice, 0);
	hipEventSynchronize(stopTimeDevice);
	hipEventElapsedTime(&elapsedTimeDevice, startTimeDevice, stopTimeDevice);
    printf("Device Elapsed Time: %f ms\n", elapsedTimeDevice);

    // Check for any errors
    int exitStatus = EXIT_SUCCESS;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << '\n';
        exitStatus = EXIT_FAILURE;
    }
    
    return exitStatus;
}
