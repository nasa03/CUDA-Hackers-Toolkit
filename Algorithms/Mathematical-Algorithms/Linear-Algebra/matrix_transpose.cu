#include "hip/hip_runtime.h"
// TODO: Finish complete program

__global__ void MatrixTranspose(float *odata, const float *idata) {
    __shared__ float tile[TILE_DIM][TILE_DIM];
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * width + x];
    }
    __syncthreads();
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        odata[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
    }
}