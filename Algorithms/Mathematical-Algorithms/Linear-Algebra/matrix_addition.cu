#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

// Define data input kernel
template <typename T>
__global__ void DataInputKernel(T *d_A, T *d_B) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i < NUM_ROW_ELEMENTS && j < NUM_COL_ELEMENTS) {
        d_A[i][j] = 2;
        d_B[i][j] = 3;
    }
}

// Define matrix addition kernel
template <typename T>
__global__ void MatrixAdditionKernel(T *d_A, T *d_B, T *d_C) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    if (i < NUM_ROW_ELEMENTS && j < NUM_COL_ELEMENTS) {
        d_C[i][j] = d_A[i][j] + d_B[i][j];
    }
}

int main() {
    // Define input data parameters
    const int NUM_ROW_ELEMENTS = 1 << 5;
    const int NUM_COL_ELEMENTS = 1 << 5;
    const int TOTAL_SIZE = NUM_ROW_ELEMENTS * NUM_COL_ELEMENTS * sizeof(int);

    // Define execution configuration variables
    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocksPerGrid(std::ceil(NUM_ROW_ELEMENTS / 32.0), std::ceil(NUM_COL_ELEMENTS / 32.0));

    // Declare pointers to input data on device
    int(*d_A)[NUM_COL_ELEMENTS], (*d_B)[NUM_COL_ELEMENTS];

    // Declare pointers to output data on both host and device
    int(*h_C)[NUM_COL_ELEMENTS], (*d_C)[NUM_COL_ELEMENTS];

    // Allocate device memory
    hipMalloc((void **) &d_A, TOTAL_SIZE);
    hipMalloc((void **) &d_B, TOTAL_SIZE);
    hipMalloc((void **) &d_C, TOTAL_SIZE);

    // Launch data input kernel on device
    DataInputKernel <<<numBlocksPerGrid, numThreadsPerBlock>>> (d_A, d_B);

    // Launch matrix addition kernel on device
    MatrixAdditionKernel <<<numBlocksPerGrid, numThreadsPerBlock>>> (d_A, d_B, d_C);

    // Wait for the device to finish computing
    hipDeviceSynchronize();

    // Transfer output data from device to host
    hipMemcpy(h_C, d_C, TOTAL_SIZE, hipMemcpyDeviceToHost);

    // Print output data
    printf("%d ", h_C);
    for (int i = 0; i < NUM_ROW_ELEMENTS; ++i) {
        for (int j = 0; j < NUM_COL_ELEMENTS; ++j) {
            std::cout << h_C[i][j] << ' ';
        }
    }
    std::cout << '\n';

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Check for errors
    int exitStatus = EXIT_SUCCESS;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << '\n';
        exitStatus = EXIT_FAILURE;
    }

    return exitStatus;
}