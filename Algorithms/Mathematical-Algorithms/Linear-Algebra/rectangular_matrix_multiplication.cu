
#include <hip/hip_runtime.h>
#include <iostream>

// Define host constants
constexpr unsigned BLOCK_DIM = 1 << 5;
constexpr unsigned NUM_ROWS_A = 1 << 7;
constexpr unsigned SHARED_DIM = 1 << 8;
constexpr unsigned NUM_COLS_B = 1 << 9;
constexpr unsigned NUM_STREAMS = 1 << 1;
constexpr unsigned TOTAL_SIZE_A = NUM_ROWS_A * SHARED_DIM;
constexpr unsigned TOTAL_SIZE_B = SHARED_DIM * NUM_COLS_B;
constexpr unsigned TOTAL_SIZE_C = NUM_ROWS_A * NUM_COLS_B;
constexpr unsigned TOTAL_PITCH_A = TOTAL_SIZE_A * sizeof(float);
constexpr unsigned TOTAL_PITCH_B = TOTAL_SIZE_B * sizeof(float);
constexpr unsigned TOTAL_PITCH_C = TOTAL_SIZE_C * sizeof(float);

// Define device constants
__constant__ unsigned DEVICE_NUM_ROWS_A = NUM_ROWS_A;
__constant__ unsigned DEVICE_SHARED_DIM = SHARED_DIM;
__constant__ unsigned DEVICE_NUM_COLS_B = NUM_COLS_B;

// Define rectangular matrix multiplication kernel
__global__ void RectangularMatrixMultiplicationKernel(float *A, float *B, float *C) {
    unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned col = blockIdx.x * blockDim.x + threadIdx.x;
    float result = 0.0f;
    if (row < DEVICE_NUM_ROWS_A && col < DEVICE_NUM_COLS_B) {
        for (unsigned i = 0; i < DEVICE_SHARED_DIM; ++i) {
            result += A[row * DEVICE_SHARED_DIM + i] * B[i * DEVICE_NUM_COLS_B + col];
        }
        C[row * DEVICE_NUM_COLS_B + col] = result;
    }
}

int main() {
    // Declare pointers to input and output data on host
	float *hostA = nullptr, *hostB = nullptr, *hostC = nullptr;

    // Allocate pinned host memory for input data
    hipHostMalloc((void **) &hostA, TOTAL_PITCH_A, hipHostMallocDefault);
    hipHostMalloc((void **) &hostB, TOTAL_PITCH_B, hipHostMallocDefault);
    hipHostMalloc((void **) &hostC, TOTAL_PITCH_C, hipHostMallocDefault);

    // Initialize input data on host
    for (unsigned i = 0; i < NUM_ROWS_A; ++i) {
        for (unsigned j = 0; j < SHARED_DIM; ++j) {
            hostA[i * SHARED_DIM + j] = 2.0f;
        }
    }
    for (unsigned i = 0; i < SHARED_DIM; ++i) {
        for (unsigned j = 0; j < NUM_COLS_B; ++j) {
            hostB[i * NUM_COLS_B + j] = 3.0f;
        }
    }
    
    // Declare pointers to input and output data on device
    float *deviceA = nullptr, *deviceB = nullptr, *deviceC = nullptr;

    // Allocate device memory for input and output data
    hipMalloc((void **) &deviceA, TOTAL_PITCH_A);
    hipMalloc((void **) &deviceB, TOTAL_PITCH_B);
    hipMalloc((void **) &deviceC, TOTAL_PITCH_C);

    // Copy input data from host to device
    hipMemcpy(deviceA, hostA, TOTAL_PITCH_A, hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, TOTAL_PITCH_B, hipMemcpyHostToDevice);

    // Declare event variables to measure execution time
    float elapsedTime_1, elapsedTime_2;
    hipEvent_t startTime_1, startTime_2, endTime_1, endTime_2;

    // Create events to measure execution time
    hipEventCreate(&startTime_1);
    hipEventCreate(&startTime_2);
    hipEventCreate(&endTime_1);
    hipEventCreate(&endTime_2);

    // Define kernel configuration variables
    dim3 blockDim(BLOCK_DIM, BLOCK_DIM);
    dim3 gridDim((NUM_ROWS_A - 1) / blockDim.x + 1, (NUM_COLS_B - 1) / blockDim.y + 1);

    // Launch rectangular matrix multiplication kernels on device and record start of execution
    RectangularMatrixMultiplicationKernel<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC);
    hipEventRecord(startTime_1, 0);
    RectangularMatrixMultiplicationKernel<<<gridDim, blockDim>>>(deviceA, deviceB, deviceC);
    hipEventRecord(startTime_2, 0);
    
    // Synchronize start of execution calls
    hipEventSynchronize(startTime_1);
    hipEventSynchronize(startTime_2);

    // Record end of execution
    hipEventRecord(endTime_1, 0);
    hipEventRecord(endTime_2, 0);

    // Synchronize end of execution calls
    hipEventSynchronize(endTime_1);
    hipEventSynchronize(endTime_2);

    // Calculate and print elapsed time
    hipEventElapsedTime(&elapsedTime_1, startTime_1, endTime_1);
    hipEventElapsedTime(&elapsedTime_2, startTime_2, endTime_2);
    std::cout << "Elapsed Time on Device Stream №1: " << elapsedTime_1 << " ms\n";
    std::cout << "Elapsed Time on Device Stream №2: " << elapsedTime_2 << " ms\n";

    // Destroy events
    hipEventDestroy(startTime_1);
    hipEventDestroy(startTime_2);
    hipEventDestroy(endTime_1);
    hipEventDestroy(endTime_2);

    // Transfer output data from device to host
    hipMemcpy(hostC, deviceC, TOTAL_PITCH_C, hipMemcpyDeviceToHost);

    // Print output data on host
    std::cout << "C = A x B:\n";
    for (unsigned i = 0; i < NUM_ROWS_A; ++i) {
        for (unsigned j = 0; j < NUM_COLS_B; ++j) {
            std::cout << hostC[i * NUM_COLS_B + j] << ' ';
        }
        std::cout << '\n';
    }

    // Free device memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    // Free pinned host memory
    hipHostFree(hostA);
    hipHostFree(hostB);
    hipHostFree(hostC);

    // Check for errors
    unsigned exitStatus = EXIT_SUCCESS;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << '\n';
        exitStatus = EXIT_FAILURE;
    }

    return exitStatus;
}