
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipfft/hipfft.h>

// Rename float2 type to complex number
typedef float2 Complex;

// Define global constants in host memory
constexpr unsigned BLOCK_DIM = 1 << 3;
constexpr unsigned SIGNAL_LENGTH = 1 << 5;
constexpr unsigned FILTER_LENGTH = 1 << 3;
constexpr unsigned FIRST_HALF_FILTER_LENGTH = FILTER_LENGTH / 2;
constexpr unsigned SIGNAL_BYTES = SIGNAL_LENGTH * sizeof(Complex);
constexpr unsigned FILTER_BYTES = FILTER_LENGTH * sizeof(Complex);
constexpr unsigned SECOND_HALF_FILTER_LENGTH = FILTER_LENGTH - FIRST_HALF_FILTER_LENGTH;
constexpr unsigned PADDED_INPUT_DATA_LENGTH = SIGNAL_LENGTH + SECOND_HALF_FILTER_LENGTH;
constexpr unsigned PADDED_INPUT_DATA_BYTES = PADDED_INPUT_DATA_LENGTH * sizeof(Complex);
constexpr unsigned FIRST_HALF_FILTER_BYTES = FIRST_HALF_FILTER_LENGTH * sizeof(Complex);
constexpr unsigned SECOND_HALF_FILTER_BYTES = SECOND_HALF_FILTER_LENGTH * sizeof(Complex);

// Define operations on complex numbers
__device__ Complex ComplexScaling(Complex a, float s)
{
    Complex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

__host__ __device__ Complex ComplexAddition(Complex a, Complex b)
{
    Complex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

__host__ __device__ Complex ComplexMultiplication(Complex a, Complex b)
{
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

__global__ void ComplexMultiplicationAndScaling(Complex *a, const Complex *b)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < PADDED_INPUT_DATA_LENGTH; i += numThreads) {
        a[i] = ComplexScaling(ComplexMultiplication(a[i], b[i]), 1.0f / PADDED_INPUT_DATA_LENGTH);
    }
}

// Define custom 1D FFT convolution calculation kernel
__global__ void CustomConvolutionKernel(const Complex *signal, const Complex *filter, Complex *filteredSignal) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ Complex tile[BLOCK_DIM];
    tile[threadIdx.x] = signal[i];
    __syncthreads();
    Complex signalValue, filteredValue;
    filteredValue.x = 0.0f;
    filteredValue.y = 0.0f;
    int start = i - FIRST_HALF_FILTER_LENGTH;
    for (int j = 0; j < FILTER_LENGTH; ++j) {
        if (start + j >= 0 && start + j < SIGNAL_LENGTH) {
            signalValue = (start + j >= blockIdx.x * blockDim.x && start + j < (blockIdx.x + 1) * blockDim.x)
                          ? (tile[threadIdx.x + j - FIRST_HALF_FILTER_LENGTH])
                          : (signal[start + j]);
            filteredValue = ComplexAddition(filteredValue, ComplexMultiplication(signalValue, filter[j]));
        }
    }
    filteredSignal[i] = filteredValue;
}


int main() {
    std::cout << "CUSTOM DEVICE KERNEL EXECUTION\n";

    // Declare pointers to input and output data on host
    Complex *hostFilter = nullptr;
    Complex *hostSignal = nullptr;
    Complex *hostFilteredSignal = nullptr;

    // Declare pointers to input and output data on device
    Complex *deviceFilter = nullptr;
    Complex *deviceSignal = nullptr;
    Complex *deviceFilteredSignal = nullptr;

    // Allocate pinned host memory for input and output data
    hipHostMalloc((void **) &hostSignal, PADDED_INPUT_DATA_BYTES, hipHostMallocDefault);
    hipHostMalloc((void **) &hostFilter, PADDED_INPUT_DATA_BYTES, hipHostMallocDefault);
    hipHostMalloc((void **) &hostFilteredSignal, PADDED_INPUT_DATA_BYTES, hipHostMallocDefault);

    // Allocate device memory for input and output data
    hipMalloc((void **) &deviceSignal, PADDED_INPUT_DATA_BYTES);
    hipMalloc((void **) &deviceFilter, PADDED_INPUT_DATA_BYTES);
    hipMalloc((void **) &deviceFilteredSignal, PADDED_INPUT_DATA_BYTES);

    // Assign signal data on host
    for (unsigned i = 0; i < SIGNAL_LENGTH; ++i) {
        hostSignal[i].x = rand() % RAND_MAX;
        hostSignal[i].y = rand() % RAND_MAX;
    }

    // Assign filter data on host
    for (unsigned j = 0; j < FILTER_LENGTH; ++j) {
        hostFilter[j].x = rand() % RAND_MAX;
        hostFilter[j].y = rand() % RAND_MAX;
    }

    // Pad signal data on host
    hipMemset(hostSignal + SIGNAL_LENGTH, 0, PADDED_INPUT_DATA_BYTES - SIGNAL_BYTES);
    
    // Pad filter data on host
    Complex *hostFilterCopy = nullptr;
    hipHostMalloc((void **) &hostFilterCopy, PADDED_INPUT_DATA_BYTES, hipHostMallocDefault);
    hipMemcpy(hostFilterCopy, hostFilter + FIRST_HALF_FILTER_LENGTH, SECOND_HALF_FILTER_BYTES, hipMemcpyHostToHost);
    hipMemset(hostFilterCopy + SECOND_HALF_FILTER_LENGTH, 0, PADDED_INPUT_DATA_BYTES - FILTER_BYTES);
    hipMemcpy(hostFilterCopy + PADDED_INPUT_DATA_LENGTH - FIRST_HALF_FILTER_LENGTH, hostFilter, FIRST_HALF_FILTER_BYTES, hipMemcpyHostToHost);
    hostFilter = hostFilterCopy;

    // Copy padded input data from host to device
    hipMemcpy(deviceSignal, hostSignal, PADDED_INPUT_DATA_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(deviceFilter, hostFilter, PADDED_INPUT_DATA_BYTES, hipMemcpyHostToDevice);

    // Declare event variables to measure execution time
    float elapsedTime;
    hipEvent_t startTime, endTime;

    // Create events to measure execution time
    hipEventCreate(&startTime);
    hipEventCreate(&endTime);

    // Define kernel configuration variables
    dim3 blockDim(BLOCK_DIM);
    dim3 gridDim((PADDED_INPUT_DATA_LENGTH - 1) / BLOCK_DIM + 1);

    // Launch custom 1D FFT convolution calculation kernel on device and record start of execution
    CustomConvolutionKernel<<<gridDim, blockDim>>>(deviceSignal, deviceFilter, deviceFilteredSignal);

    // Record start of execution
    hipEventRecord(startTime, 0);
    
    // Synchronize start of execution call
    hipEventSynchronize(startTime);

    // Record end of execution
    hipEventRecord(endTime, 0);

    // Synchronize end of execution call
    hipEventSynchronize(endTime);

    // Transfer output data from device to host
    hipMemcpy(hostFilteredSignal, deviceFilteredSignal, PADDED_INPUT_DATA_BYTES, hipMemcpyDeviceToHost);

    // Print output data on host
    std::cout << "Filtered Signal:\n";
    for (unsigned i = 0; i < PADDED_INPUT_DATA_LENGTH; ++i) {
        std::cout << hostFilteredSignal[i].x << ' ' << hostFilteredSignal[i].y << '\n';
    }
    std::cout << '\n';

    // Calculate and print elapsed time
    hipEventElapsedTime(&elapsedTime, startTime, endTime);
    std::cout << "Elapsed Time on Device: " << elapsedTime << " ms\n";

    // Destroy events
    hipEventDestroy(startTime);
    hipEventDestroy(endTime);

    std::cout << "\nLIBRARY DEVICE KERNEL EXECUTION\n";

    // Create computation plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, PADDED_INPUT_DATA_LENGTH, HIPFFT_C2C, 1);

    // Create events to measure execution time
    hipEventCreate(&startTime);
    hipEventCreate(&endTime);

    // Apply forward Discrete Fourier Transform to input data on device
    hipfftExecC2C(plan, (hipfftComplex *)deviceSignal, (hipfftComplex *)deviceSignal, HIPFFT_FORWARD);
    hipfftExecC2C(plan, (hipfftComplex *)deviceFilter, (hipfftComplex *)deviceFilter, HIPFFT_FORWARD);

    // Multiply and normalize the complex frequency coefficients on device
    ComplexMultiplicationAndScaling<<<gridDim, blockDim>>>(deviceSignal, deviceFilter);

    // Apply inverse Discrete Fourier Transform to input data on device
    hipfftExecC2C(plan, (hipfftComplex *)deviceSignal, (hipfftComplex *)deviceSignal, HIPFFT_BACKWARD);

    // Record start of execution
    hipEventRecord(startTime, 0);
    
    // Synchronize start of execution call
    hipEventSynchronize(startTime);

    // Record end of execution
    hipEventRecord(endTime, 0);

    // Synchronize end of execution call
    hipEventSynchronize(endTime);

    // Transfer output data from device to host
    hipMemcpy(hostFilteredSignal, deviceSignal, PADDED_INPUT_DATA_BYTES, hipMemcpyDeviceToHost);

    // Print output data on host
    std::cout << "Filtered Signal:\n";
    for (unsigned i = 0; i < PADDED_INPUT_DATA_LENGTH; ++i) {
        std::cout << hostFilteredSignal[i].x << ' ' << hostFilteredSignal[i].y << '\n';
    }
    std::cout << '\n';

    // Calculate and print elapsed time
    hipEventElapsedTime(&elapsedTime, startTime, endTime);
    std::cout << "Elapsed Time on Device: " << elapsedTime << " ms\n";

    // Destroy events
    hipEventDestroy(startTime);
    hipEventDestroy(endTime);

    // Destroy computation plan
    hipfftDestroy(plan);

    // Free device memory
    hipFree(deviceSignal);
    hipFree(deviceFilter);
    hipFree(deviceFilteredSignal);

    // Free pinned host memory
    hipHostFree(hostSignal);
    hipHostFree(hostFilter);
    hipHostFree(hostFilteredSignal);

    // Check for errors
    unsigned exitStatus = EXIT_SUCCESS;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << '\n';
        exitStatus = EXIT_FAILURE;
    }
    
    return exitStatus;
}