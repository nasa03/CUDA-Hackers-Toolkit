
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <array>

// Rename float2 type to complex number
typedef float2 Complex;

// Define global constants in host memory
constexpr unsigned BLOCK_DIM = 1 << 5;
constexpr unsigned SIGNAL_LENGTH = 1 << 13;
constexpr unsigned FILTER_LENGTH = 1 << 5;
constexpr unsigned FIRST_HALF_FILTER_LENGTH = FILTER_LENGTH / 2;
constexpr unsigned FILTER_PADDING_LENGTH = SIGNAL_LENGTH - FIRST_HALF_FILTER_LENGTH;
constexpr unsigned SECOND_HALF_FILTER_LENGTH = FILTER_LENGTH - FIRST_HALF_FILTER_LENGTH;
constexpr unsigned PADDED_INPUT_DATA_LENGTH = SIGNAL_LENGTH + SECOND_HALF_FILTER_LENGTH;
constexpr unsigned PADDED_INPUT_DATA_BYTES = PADDED_INPUT_DATA_LENGTH * sizeof(Complex);

// Define operations on complex numbers
__device__ Complex ComplexScaling(Complex a, float s)
{
    Complex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

__host__ __device__ Complex ComplexAddition(Complex a, Complex b)
{
    Complex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

__host__ __device__ Complex ComplexMultiplication(Complex a, Complex b)
{
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

__global__ void ComplexMultiplicationAndScaling(Complex *a, const Complex *b)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < PADDED_INPUT_DATA_LENGTH; i += numThreads) {
        a[i] = ComplexScaling(ComplexMultiplication(a[i], b[i]), 1.0f / PADDED_INPUT_DATA_LENGTH);
    }
}

// Define custom 1D FFT convolution calculation kernel
__global__ void CustomConvolutionKernel(const Complex *signal, const Complex *filter, Complex *filteredSignal) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ Complex tile[BLOCK_DIM];
    tile[threadIdx.x] = signal[i];
    __syncthreads();
    Complex signalValue, filteredValue;
    filteredValue.x = 0.0f;
    filteredValue.y = 0.0f;
    int start = i - FIRST_HALF_FILTER_LENGTH;
    for (int j = 0; j < FILTER_LENGTH; ++j) {
        if (start + j >= 0 && start + j < SIGNAL_LENGTH) {
            signalValue = (start + j >= blockIdx.x * blockDim.x && start + j < (blockIdx.x + 1) * blockDim.x)
                          ? (tile[threadIdx.x + j - FIRST_HALF_FILTER_LENGTH])
                          : (signal[start + j]);
            filteredValue = ComplexAddition(filteredValue, ComplexMultiplication(signalValue, filter[j]));
        }
    }
    filteredSignal[i] = filteredValue;
}


int main() {
    std::cout << "CUSTOM DEVICE KERNEL EXECUTION\n";

    // Declare pointers to input and output data on host
    Complex *hostFilter = nullptr;
    Complex *hostSignal = nullptr;
    Complex *hostFilteredSignal = nullptr;

    // Declare pointers to input and output data on device
    Complex *deviceFilter = nullptr;
    Complex *deviceSignal = nullptr;
    Complex *deviceFilteredSignal = nullptr;

    // Allocate pinned host memory for input and output data
    hipHostMalloc((void **) &hostSignal, PADDED_INPUT_DATA_BYTES, hipHostMallocDefault);
    hipHostMalloc((void **) &hostFilter, PADDED_INPUT_DATA_BYTES, hipHostMallocDefault);
    hipHostMalloc((void **) &hostFilteredSignal, PADDED_INPUT_DATA_BYTES, hipHostMallocDefault);

    // Allocate device memory for input and output data
    hipMalloc((void **) &deviceSignal, PADDED_INPUT_DATA_BYTES);
    hipMalloc((void **) &deviceFilter, PADDED_INPUT_DATA_BYTES);
    hipMalloc((void **) &deviceFilteredSignal, PADDED_INPUT_DATA_BYTES);

    // Assign signal data on host
    for (unsigned i = 0; i < SIGNAL_LENGTH; ++i) {
        hostSignal[i].x = rand() % RAND_MAX;
        hostSignal[i].y = rand() % RAND_MAX;
    }

    // Assign filter data on host
    for (unsigned j = 0; j < FILTER_LENGTH; ++j) {
        hostFilter[j].x = rand() % RAND_MAX;
        hostFilter[j].y = rand() % RAND_MAX;
    }

    // Pad signal data on host
    for (unsigned i = SIGNAL_LENGTH; i < PADDED_INPUT_DATA_LENGTH; ++i) {
        hostSignal[i].x = 0.0f;
        hostSignal[i].y = 0.0f;
    }
    
    // Pad filter data on host
    std::array<Complex, PADDED_INPUT_DATA_BYTES> hostFilterCopy;
    for (unsigned j = 0; j < FILTER_LENGTH; ++j) {
        hostFilterCopy[j].x = hostFilter[j].x;
        hostFilterCopy[j].y = hostFilter[j].y;
    }
    for (unsigned j = FIRST_HALF_FILTER_LENGTH, jCopy = 0; j < FILTER_LENGTH; ++j, ++jCopy) {
        hostFilterCopy[jCopy] = hostFilter[j];
    }
    for (unsigned jCopy = SECOND_HALF_FILTER_LENGTH, k = 0; k < FILTER_PADDING_LENGTH; ++jCopy, k++) {
        hostFilterCopy[jCopy].x = 0.0f;
        hostFilterCopy[jCopy].y = 0.0f;
    }
    for (unsigned j = 0, jCopy = PADDED_INPUT_DATA_LENGTH - FIRST_HALF_FILTER_LENGTH; j < FIRST_HALF_FILTER_LENGTH; ++j, ++jCopy) {
        hostFilterCopy[jCopy] = hostFilter[j];
    }
    for (unsigned j = 0; j < PADDED_INPUT_DATA_BYTES; ++j) {
        hostFilter[j].x = hostFilterCopy[j].x;
        hostFilter[j].y = hostFilterCopy[j].y;
    }

    // Copy padded input data from host to device
    hipMemcpy(deviceSignal, hostSignal, PADDED_INPUT_DATA_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(deviceFilter, hostFilter, PADDED_INPUT_DATA_BYTES, hipMemcpyHostToDevice);

    // Declare event variables to measure execution time
    float elapsedTime;
    hipEvent_t startTime, endTime;

    // Create events to measure execution time
    hipEventCreate(&startTime);
    hipEventCreate(&endTime);

    // Define kernel configuration variables
    dim3 blockDim(BLOCK_DIM);
    dim3 gridDim((PADDED_INPUT_DATA_LENGTH - 1) / BLOCK_DIM + 1);

    // Launch custom 1D FFT convolution calculation kernel on device and record start of execution
    CustomConvolutionKernel<<<gridDim, blockDim>>>(deviceSignal, deviceFilter, deviceFilteredSignal);

    // Record start of execution
    hipEventRecord(startTime, 0);
    
    // Synchronize start of execution call
    hipEventSynchronize(startTime);

    // Record end of execution
    hipEventRecord(endTime, 0);

    // Synchronize end of execution call
    hipEventSynchronize(endTime);

    // Transfer output data from device to host
    hipMemcpy(hostFilteredSignal, deviceFilteredSignal, PADDED_INPUT_DATA_BYTES, hipMemcpyDeviceToHost);

    // Print output data on host
    std::cout << "Filtered Signal:\n";
    for (unsigned i = 0; i < PADDED_INPUT_DATA_LENGTH; ++i) {
        std::cout << hostFilteredSignal[i].x << ' ' << hostFilteredSignal[i].y << '\n';
    }
    std::cout << '\n';

    // Calculate and print elapsed time
    hipEventElapsedTime(&elapsedTime, startTime, endTime);
    std::cout << "Elapsed Time on Device: " << elapsedTime << " ms\n";

    // Destroy events
    hipEventDestroy(startTime);
    hipEventDestroy(endTime);

    std::cout << "\nLIBRARY DEVICE KERNEL EXECUTION\n";

    // Create computation plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, PADDED_INPUT_DATA_LENGTH, HIPFFT_C2C, 1);

    // Create events to measure execution time
    hipEventCreate(&startTime);
    hipEventCreate(&endTime);

    // Apply forward Discrete Fourier Transform to input data on device
    hipfftExecC2C(plan, (hipfftComplex *)deviceSignal, (hipfftComplex *)deviceSignal, HIPFFT_FORWARD);
    hipfftExecC2C(plan, (hipfftComplex *)deviceFilter, (hipfftComplex *)deviceFilter, HIPFFT_FORWARD);

    // Multiply and normalize the complex frequency coefficients on device
    ComplexMultiplicationAndScaling<<<gridDim, blockDim>>>(deviceSignal, deviceFilter);

    // Apply inverse Discrete Fourier Transform to input data on device
    hipfftExecC2C(plan, (hipfftComplex *)deviceSignal, (hipfftComplex *)deviceSignal, HIPFFT_BACKWARD);

    // Record start of execution
    hipEventRecord(startTime, 0);
    
    // Synchronize start of execution call
    hipEventSynchronize(startTime);

    // Record end of execution
    hipEventRecord(endTime, 0);

    // Synchronize end of execution call
    hipEventSynchronize(endTime);

    // Transfer output data from device to host
    hipMemcpy(hostFilteredSignal, deviceSignal, PADDED_INPUT_DATA_BYTES, hipMemcpyDeviceToHost);

    // Print output data on host
    std::cout << "Filtered Signal:\n";
    for (unsigned i = 0; i < PADDED_INPUT_DATA_LENGTH; ++i) {
        std::cout << hostFilteredSignal[i].x << ' ' << hostFilteredSignal[i].y << '\n';
    }
    std::cout << '\n';

    // Calculate and print elapsed time
    hipEventElapsedTime(&elapsedTime, startTime, endTime);
    std::cout << "Elapsed Time on Device: " << elapsedTime << " ms\n";

    // Destroy events
    hipEventDestroy(startTime);
    hipEventDestroy(endTime);

    // Destroy computation plan
    hipfftDestroy(plan);

    // Free device memory
    hipFree(deviceSignal);
    hipFree(deviceFilter);
    hipFree(deviceFilteredSignal);

    // Free pinned host memory
    hipHostFree(hostSignal);
    hipHostFree(hostFilter);
    hipHostFree(hostFilteredSignal);

    // Check for errors
    unsigned exitStatus = EXIT_SUCCESS;
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Error: " << hipGetErrorString(err) << '\n';
        exitStatus = EXIT_FAILURE;
    }
    
    return exitStatus;
}